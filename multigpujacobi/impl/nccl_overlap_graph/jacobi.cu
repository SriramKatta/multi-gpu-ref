#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <math.h>
#include <nvtx3/nvtx3.hpp>
#include <nccl.h>

#include "utils.h"

#ifndef SKIP_CUDA_AWARENESS_CHECK
#include <mpi-ext.h>
#if !defined(MPIX_CUDA_AWARE_SUPPORT) || !MPIX_CUDA_AWARE_SUPPORT
#error "The used MPI Implementation does not have CUDA-aware support or CUDA-aware support can't be determined. Define SKIP_CUDA_AWARENESS_CHECK to skip this check."
#endif
#endif


using real = double;
#define NCCL_REAL_TYPE ncclDouble

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int N, const int my_ny);
void launch_initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                  const real pi, const int offset, const int N, const int my_ny);

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                              const int iy_start, const int iy_end, const int N);
void launch_jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                          const int iy_start, const int iy_end, const int N, hipStream_t stream);
void Halo_exchange(real *a, real *a_new, int N, const int top, int iy_end, const int bottom, int iy_start, ncclComm_t, hipStream_t);

int main(int argc, char *argv[])
{
  MPI_CALL(MPI_Init(&argc, &argv));
  int rank = 0, nranks = 1;
  CUDA_CALL(hipGetDeviceCount(&nranks));
  MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
  MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &nranks));

  CUDA_CALL(hipSetDevice(rank % nranks));
  CUDA_CALL(hipFree(0));

  ncclComm_t ncclcomm;
  ncclUniqueId id;
  if (rank == 0)
    NCCL_CALL(ncclGetUniqueId(&id));
  MPI_CALL(MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  NCCL_CALL(ncclCommInitRank(&ncclcomm, nranks, id, rank));
  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));

  int N = 1024;
  if (argc > 1)
  {
    N = atoi(argv[1]);
    if (N % 1024 != 0)
    {
      if (rank == 0)
        printf("size should be a multiple of 1024\n");
      MPI_CALL(MPI_Finalize());
      exit(EXIT_SUCCESS);
    }
  }

  int chunk_size = rowsinrank(rank, nranks, N);
  real *a, *a_new;
  CUDA_CALL(hipMalloc(&a, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMemset(a, 0, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMalloc(&a_new, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMemset(a_new, 0, N * (chunk_size + 2) * sizeof(real)));

  int iy_start_global = startrow(rank, nranks, N);
  int iy_start = 1;
  int iy_end = iy_start + chunk_size;

  // Streams with different priorities
  int lowp, highp;
  hipStream_t inner_stream, edge_stream;
  CUDA_CALL(hipDeviceGetStreamPriorityRange(&lowp, &highp));
  CUDA_CALL(hipStreamCreateWithPriority(&inner_stream, hipStreamDefault, lowp));
  CUDA_CALL(hipStreamCreateWithPriority(&edge_stream, hipStreamDefault, highp));
  hipEvent_t inner_done, edge_done;
  CUDA_CALL(hipEventCreateWithFlags(&inner_done, hipEventDisableTiming));
  CUDA_CALL(hipEventCreateWithFlags(&edge_done, hipEventDisableTiming));

  const int top_pe = (rank + 1) % nranks;
  const int bot_pe = (rank + nranks - 1) % nranks;

  // Warmup NCCL+halo exchanges
  nvtxRangePushA("NCCL_Warmup");

  for (int i = 0; i < number_of_warmups; ++i)
  {
    Halo_exchange(a_new, a, N, top_pe, iy_end, bot_pe, iy_start, ncclcomm, edge_stream);
    std::swap(a, a_new);
  }
  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipDeviceSynchronize());
  nvtxRangePop();

  hipGraph_t graphs[2];
  nvtxRangePushA("Graph_create");
  for (int g = 0; g < 2; ++g)
  {
    // Begin global capture on both streams
    CUDA_CALL(hipStreamBeginCapture(inner_stream, hipStreamCaptureModeGlobal));
    CUDA_CALL(hipStreamBeginCapture(edge_stream, hipStreamCaptureModeGlobal));

    // Launch edge-row Jacobi on edge_stream
    launch_jacobi_kernel(a_new, a, iy_start, iy_start + 1, N, edge_stream);
    launch_jacobi_kernel(a_new, a, iy_end - 1, iy_end, N, edge_stream);

    // NCCL halo exchange on edge_stream
    NCCL_CALL(ncclGroupStart());
    NCCL_CALL(ncclRecv(a_new, N, NCCL_REAL_TYPE, top_pe, ncclcomm, edge_stream));
    NCCL_CALL(ncclSend(a_new + (iy_end - 1) * N, N, NCCL_REAL_TYPE, bot_pe, ncclcomm, edge_stream));
    NCCL_CALL(ncclRecv(a_new + iy_end * N, N, NCCL_REAL_TYPE, bot_pe, ncclcomm, edge_stream));
    NCCL_CALL(ncclSend(a_new + iy_start * N, N, NCCL_REAL_TYPE, top_pe, ncclcomm, edge_stream));
    NCCL_CALL(ncclGroupEnd());

    // Inner Jacobi on inner_stream
    launch_jacobi_kernel(a_new, a, iy_start + 1, iy_end - 1, N, inner_stream);

    // Capture end: first end on edge_stream to get full graph, then on inner_stream
    CUDA_CALL(hipStreamEndCapture(edge_stream, &graphs[g]));
    CUDA_CALL(hipStreamEndCapture(inner_stream, nullptr));

    std::swap(a, a_new);
  }
  nvtxRangePop();

  // Instantiate graphs

  hipGraphExec_t graph_exec[2];
  nvtxRangePushA("Graph_init");
  for (int g = 0; g < 2; ++g)
  {
    CUDA_CALL(hipGraphInstantiateWithFlags(&graph_exec[g], graphs[g], hipGraphInstantiateFlagUseNodePriority));
    CUDA_CALL(hipGraphDestroy(graphs[g]));
  }
  nvtxRangePop();

  // Warmup graph launches
  nvtxRangePushA("Graph_Warmup");
  for (int i = 0; i < 10; ++i)
  {
    CUDA_CALL(hipGraphLaunch(graph_exec[0], inner_stream));
    CUDA_CALL(hipGraphLaunch(graph_exec[1], inner_stream));
    CUDA_CALL(hipStreamSynchronize(inner_stream));
  }
  nvtxRangePop();

  // Initialize boundaries
  CUDA_CALL(hipMemset(a, 0, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMemset(a_new, 0, N * (chunk_size + 2) * sizeof(real)));
  launch_initialize_boundaries(a, a_new, M_PI, iy_start_global - 1, N, chunk_size + 2);
  CUDA_CALL(hipDeviceSynchronize());

  // Solve
  double start = MPI_Wtime();
  nvtxRangePushA("Jacobistep");
  for (int it = 0; it < maxIt; ++it)
  {
    CUDA_CALL(hipGraphLaunch(graph_exec[it % 2], inner_stream));
  }
  CUDA_CALL(hipDeviceSynchronize());
  nvtxRangePop();
  double dur = (MPI_Wtime() - start) / maxIt;
  double maxdur = 0;
  MPI_CALL(MPI_Reduce(&dur, &maxdur, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD));

  if (rank == 0)
  {
    printf("NP %3d | LUPs %12d | perf %7.3f MLUPS/s\n", nranks, (N * N), N * N / maxdur / 1e6);
  }

  // Cleanup
  CUDA_CALL(hipFree(a));
  CUDA_CALL(hipFree(a_new));
  CUDA_CALL(hipGraphExecDestroy(graph_exec[1]));
  CUDA_CALL(hipGraphExecDestroy(graph_exec[0]));
  CUDA_CALL(hipEventDestroy(inner_done));
  CUDA_CALL(hipEventDestroy(edge_done));
  CUDA_CALL(hipStreamDestroy(inner_stream));
  CUDA_CALL(hipStreamDestroy(edge_stream));
  NCCL_CALL(ncclCommDestroy(ncclcomm));
  MPI_Finalize();
  return 0;
}

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int N, const int my_ny)
{
  for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x)
  {
    const real y0 = sin(2.0 * pi * (offset + iy) / (N - 1));
    a[iy * N + 0] = y0;
    a[iy * N + (N - 1)] = y0;
    a_new[iy * N + 0] = y0;
    a_new[iy * N + (N - 1)] = y0;
  }
}

void launch_initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                  const real pi, const int offset, const int N, const int my_ny)
{
  initialize_boundaries<<<my_ny / 128 + 1, 128>>>(a_new, a, pi, offset, N, my_ny);
  CUDA_CALL(hipGetLastError());
}

void launch_jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                          const int iy_start, const int iy_end,
                          const int N, hipStream_t stream)
{
  constexpr int dim_block_x = 32;
  constexpr int dim_block_y = 32;
  dim3 thread_dim(dim_block_x, dim_block_x);
  dim3 block_dim((N + dim_block_x - 1) / dim_block_x,
                 ((iy_end - iy_start) + dim_block_y - 1) / dim_block_y);
  jacobi_kernel<dim_block_x, dim_block_y><<<block_dim, thread_dim, 0, stream>>>(
      a_new, a, iy_start, iy_end, N);
  CUDA_CALL(hipGetLastError());
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                              const int iy_start, const int iy_end, const int N)
{
  int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
  int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

  if (iy < iy_end && ix < (N - 1))
  {
    const real new_val = 0.25 * (a[iy * N + ix + 1] + a[iy * N + ix - 1] +
                                 a[(iy + 1) * N + ix] + a[(iy - 1) * N + ix]);
    a_new[iy * N + ix] = new_val;
  }
}

void Halo_exchange(real *a_new, real *a, int N, const int top, int iy_end, const int bottom, int iy_start, ncclComm_t nccl_comm, hipStream_t edge_stream)
{
  NCCL_CALL(ncclGroupStart());
  // clang-format off
  NCCL_CALL(ncclRecv(a_new                   , N, NCCL_REAL_TYPE, top   , nccl_comm, edge_stream));
  NCCL_CALL(ncclSend(a_new + (iy_end - 1) * N, N, NCCL_REAL_TYPE, bottom, nccl_comm, edge_stream));
  NCCL_CALL(ncclRecv(a_new + (iy_end * N)    , N, NCCL_REAL_TYPE, bottom, nccl_comm, edge_stream));
  NCCL_CALL(ncclSend(a_new + (iy_start * N)  , N, NCCL_REAL_TYPE, top   , nccl_comm, edge_stream));
  // clang-format on
  NCCL_CALL(ncclGroupEnd());
  CUDA_CALL(hipStreamSynchronize(edge_stream));
}
