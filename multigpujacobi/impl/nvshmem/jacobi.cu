#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <math.h>
#include <nvshmem.h>
#include <nvshmemx.h>

#include <nvtx3/nvtx3.hpp>
#include "utils.h"

// TODO: comment out on my personal machine since it doesn't have cuda aware mpi :(
#ifndef SKIP_CUDA_AWARENESS_CHECK
#include <mpi-ext.h>
#if !defined(MPIX_CUDA_AWARE_SUPPORT) || !MPIX_CUDA_AWARE_SUPPORT
#error "The used MPI Implementation does not have CUDA-aware support or CUDA-aware \
support can't be determined. Define SKIP_CUDA_AWARENESS_CHECK to skip this check."
#endif
#endif

constexpr int number_of_warmups = 10;
constexpr int maxIt = 1000;

using real = double;
#define NCCL_REAL_TYPE ncclDouble

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int N, const int my_ny);
void launch_initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                  const real pi, const int offset, const int N, const int my_ny);
template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                              const int iy_start, const int iy_end, const int N);
void launch_jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                          const int iy_start, const int iy_end, const int N, hipStream_t stream);
void Halo_exchange(real *a, real *a_new, int N, const int top, int iy_end, const int bottom, int iy_start, ncclComm_t, hipStream_t, hipStream_t);

int main(int argc, char *argv[])
{
  MPI_CALL(MPI_Init(&argc, &argv));
  int rank = 0;
  int nranks = 1;
  int num_devices = 0;
  // assumption is that we are only on single node but easily extendable using MPI_Comm_split_type(.... MPI_COMM_TYPE_SHARED...
  {
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &nranks));
  }

  // important part of cuda aware mpi
  {
    CUDA_CALL(hipGetDeviceCount(&num_devices));
    CUDA_CALL(hipSetDevice(rank % num_devices));
    CUDA_CALL(hipFree(0));
  }

  // pass the communication hadling to NCCL
  ncclComm_t ncclcomm;
  ncclUniqueId id;
  if (rank == 0)
    NCCL_CALL(ncclGetUniqueId(&id));
  MPI_CALL(MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  NCCL_CALL(ncclCommInitRank(&ncclcomm, nranks, id, rank));

  // just to be safe
  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));

  // just to simplyfy the program
  int N = 1024;
  if (argc > 1)
  {
    N = atoi(argv[1]);
    if (N % 1024 != 0)
    {
      if (rank == 0)
        printf("size should be a multiple of 1024\n");
      MPI_CALL(MPI_Finalize());
      exit(EXIT_SUCCESS);
    }
  }

  // computing local chunk size
  int chunk_size;
  int chunk_size_low = (N - 2) / nranks;
  int chunk_size_high = chunk_size_low + 1;
  int num_ranks_low = nranks * chunk_size_low + nranks - (N - 2);
  if (rank < num_ranks_low)
    chunk_size = chunk_size_low;
  else
    chunk_size = chunk_size_high;

  // setting up data for each GPU
  real *a;
  real *a_new;
  CUDA_CALL(hipMalloc(&a, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMemset(a, 0, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMalloc(&a_new, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMemset(a_new, 0, N * (chunk_size + 2) * sizeof(real)));

  // Calculate local domain boundaries
  int iy_start_global; // My start index in the global array
  if (rank < num_ranks_low)
  {
    iy_start_global = rank * chunk_size_low + 1;
  }
  else
  {
    iy_start_global =
        num_ranks_low * chunk_size_low + (rank - num_ranks_low) * chunk_size_high + 1;
  }

  int iy_start = 1;
  int iy_end = iy_start + chunk_size;

  launch_initialize_boundaries(a, a_new, M_PI, iy_start_global - 1, N, (chunk_size + 2));
  CUDA_CALL(hipDeviceSynchronize());

  int highpriority = 0, lowpriority = 0;
  hipStream_t inner_stream;
  hipStream_t top_stream;
  hipStream_t bottom_stream;
  CUDA_CALL(hipDeviceGetStreamPriorityRange(&lowpriority, &highpriority));
  CUDA_CALL(hipStreamCreateWithPriority(&inner_stream, hipStreamDefault, lowpriority));
  CUDA_CALL(hipStreamCreateWithPriority(&top_stream, hipStreamDefault, highpriority));
  CUDA_CALL(hipStreamCreateWithPriority(&bottom_stream, hipStreamDefault, highpriority));

  const int top = (rank + nranks - 1) % nranks;
  const int bottom = (rank + 1) % nranks;

  nvtxRangePushA("MPI_Warmup");
  for (size_t i = 0; i < number_of_warmups; i++)
  {
    Halo_exchange(a_new, a, N, top, iy_end, bottom, iy_start, ncclcomm, top_stream, bottom_stream);
    std::swap(a, a_new);
  }
  nvtxRangePop();
  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipDeviceSynchronize());

  double start = MPI_Wtime();
  nvtxRangePushA("Full_loop");
  for (size_t it = 0; it < maxIt; ++it)
  {
    nvtx3::scoped_range loop{"Jacobi_Step"};

    nvtxRangePushA("Apply_stencil");
    launch_jacobi_kernel(a_new, a, iy_start, iy_start + 1, N, top_stream);
    launch_jacobi_kernel(a_new, a, iy_end - 1, iy_end, N, bottom_stream);
    launch_jacobi_kernel(a_new, a, iy_start + 1, iy_end - 1, N, inner_stream);
    nvtxRangePop();

    CUDA_CALL(hipStreamSynchronize(top_stream));
    CUDA_CALL(hipStreamSynchronize(bottom_stream));
    std::swap(a, a_new);
  }
  nvtxRangePop();
  double dur = MPI_Wtime() - start;
  double maxdur = 0.0;
  MPI_CALL(MPI_Reduce(&dur, &maxdur, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD));

  if (rank == 0)
  {
    // fill the performnace computation take max time
    printf("NP %3d | LUPs %12d | perf %7.3f MLUPS/s\n", nranks, (N * N), static_cast<double>(N * N) / maxdur / 1e6);
  }

  // freeing everything
  CUDA_CALL(hipFree(a));
  CUDA_CALL(hipFree(a_new));
  CUDA_CALL(hipStreamDestroy(inner_stream));
  CUDA_CALL(hipStreamDestroy(top_stream));
  CUDA_CALL(hipStreamDestroy(bottom_stream));
  NCCL_CALL(ncclCommDestroy(ncclcomm));
  MPI_Finalize();
  return 0;
}

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int N, const int my_ny)
{
  for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x)
  {
    const real y0 = sin(2.0 * pi * (offset + iy) / (N - 1));
    a[iy * N + 0] = y0;
    a[iy * N + (N - 1)] = y0;
    a_new[iy * N + 0] = y0;
    a_new[iy * N + (N - 1)] = y0;
  }
}

void launch_initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                  const real pi, const int offset, const int N, const int my_ny)
{
  initialize_boundaries<<<my_ny / 128 + 1, 128>>>(a_new, a, pi, offset, N, my_ny);
  CUDA_CALL(hipGetLastError());
}

void launch_jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                          const int iy_start, const int iy_end,
                          const int N, hipStream_t stream)
{
  constexpr int dim_block_x = 32;
  constexpr int dim_block_y = 32;
  dim3 thread_dim(dim_block_x, dim_block_x);
  dim3 block_dim((N + dim_block_x - 1) / dim_block_x,
                 ((iy_end - iy_start) + dim_block_y - 1) / dim_block_y);
  jacobi_kernel<dim_block_x, dim_block_y><<<block_dim, thread_dim, 0, stream>>>(
      a_new, a, iy_start, iy_end, N);
  CUDA_CALL(hipGetLastError());
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                              const int iy_start, const int iy_end, const int N)
{
  int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
  int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

  if (iy < iy_end && ix < (N - 1))
  {
    const real new_val = 0.25 * (a[iy * N + ix + 1] + a[iy * N + ix - 1] +
                                 a[(iy + 1) * N + ix] + a[(iy - 1) * N + ix]);
    a_new[iy * N + ix] = new_val;
    if (iy_start == iy)
    {
      nvshmem_float_p(a_new + top_iy * nx + ix, new_val, top_pe);
    }
    if ((iy_end - 1) == iy)
    {
      nvshmem_float_p(a_new + bottom_iy * nx + ix, new_val, bottom_pe);
    }
  }
}