#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <math.h>
#include <nvtx3/nvtx3.hpp>

#include "utils.h"

// TODO: commented out since my personal machine doesn't have cuda aware mpi :(
#ifndef SKIP_CUDA_AWARENESS_CHECK
#include <mpi-ext.h>
#if !defined(MPIX_CUDA_AWARE_SUPPORT) || !MPIX_CUDA_AWARE_SUPPORT
#error "The used MPI Implementation does not have CUDA-aware support or CUDA-aware \
support can't be determined. Define SKIP_CUDA_AWARENESS_CHECK to skip this check."
#endif
#endif

constexpr int number_of_warmups = 10;
constexpr int maxIt = 1000;

using real = double;

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int N, const int my_ny);
void launch_initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                  const real pi, const int offset, const int N, const int my_ny);
template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                              const int iy_start, const int iy_end, const int N, int top_pe, int bot_pe);
void launch_jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                          const int iy_start, const int iy_end, const int N, hipStream_t stream);

int main(int argc, char *argv[])
{
  MPI_CALL(MPI_Init(&argc, &argv));
  int rank = 0;
  int nranks = 1;
  int num_devices = 0;
  // assumption is that we are only on single node but easily extendable using MPI_Comm_split_type(.... MPI_COMM_TYPE_SHARED...
  {
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &nranks));
  }
  MPI_Comm comm = MPI_COMM_WORLD;

  // all nvshmem from here
  nvshmemx_init_attr_t attr;
  attr.mpi_comm = &comm;
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

  int mype_node;
  mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  CUDA_CALL(hipSetDevice(mype_node));

  // just to simplyfy the program
  int N = 1024;
  if (argc > 1)
  {
    N = atoi(argv[1]);
    if (N % 1024 != 0)
    {
      if (rank == 0)
        printf("size should be a multiple of 1024\n");
      MPI_CALL(MPI_Finalize());
      exit(EXIT_SUCCESS);
    }
  }

  // computing local chunk size
  int chunk_size = rowsinrank(rank, nranks, N);

  // setting up data for each GPU
  real *a = (real *)nvshmem_malloc(sizeof(real) * N * (chunk_size + 2));
  real *a_new = (real *)nvshmem_malloc(sizeof(real) * N * (chunk_size + 2));
  // clang-format off
  CUDA_CALL(hipMemset(a   , 0, N * (chunk_size + 2) * sizeof(real)));
  CUDA_CALL(hipMemset(a_new,0, N * (chunk_size + 2) * sizeof(real)));
  // clang-format on

  // Calculate local domain boundaries
  int iy_start_global = startrow(rank, nranks, N);

  int iy_start = 1;
  int iy_end = iy_start + chunk_size;

  launch_initialize_boundaries(a, a_new, M_PI, iy_start_global - 1, N, (chunk_size + 2));
  CUDA_CALL(hipDeviceSynchronize());

  hipStream_t compute_stream;
  CUDA_CALL(hipStreamCreate(&compute_stream));

  const int top_pe = (rank + 1) % nranks;
  const int bot_pe = (rank + nranks - 1) % nranks;

  MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));
  CUDA_CALL(hipDeviceSynchronize());

  double start = MPI_Wtime();
  for (size_t it = 0; it < maxIt; ++it)
  {
    nvtx3::scoped_range loop{"Jacobi_Step"};

    launch_jacobi_kernel(a_new, a, iy_start, iy_end, N, compute_stream);

    std::swap(a, a_new);
  }
  CUDA_CALL(hipDeviceSynchronize());
  double dur = (MPI_Wtime() - start) / maxIt;
  double maxdur = 0.0;
  MPI_CALL(MPI_Reduce(&dur, &maxdur, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD));

  if (rank == 0)
  {
    printf("NP %3d | LUPs %12d | perf %7.3f MLUPS/s\n", nranks, (N * N), static_cast<double>(N * N) / maxdur / 1e6);
  }

  // freeing everything
  nvshmem_free(a);
  nvshmem_free(a_new);
  CUDA_CALL(hipStreamDestroy(compute_stream));
  nvshmem_finalize();
  MPI_Finalize();
  return 0;
}

__global__ void initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                      const real pi, const int offset, const int N, const int my_ny)
{
  for (int iy = blockIdx.x * blockDim.x + threadIdx.x; iy < my_ny; iy += blockDim.x * gridDim.x)
  {
    const real y0 = sin(2.0 * pi * (offset + iy) / (N - 1));
    a[iy * N + 0] = y0;
    a[iy * N + (N - 1)] = y0;
    a_new[iy * N + 0] = y0;
    a_new[iy * N + (N - 1)] = y0;
  }
}

void launch_initialize_boundaries(real *__restrict__ const a_new, real *__restrict__ const a,
                                  const real pi, const int offset, const int N, const int my_ny)
{
  initialize_boundaries<<<my_ny / 128 + 1, 128>>>(a_new, a, pi, offset, N, my_ny);
  CUDA_CALL(hipGetLastError());
}

void launch_jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                          const int iy_start, const int iy_end,
                          const int N, hipStream_t stream)
{
  int mype = nvshmem_my_pe();
  int npes = nvshmem_n_pes();
  int top_pe = (mype + 1) % npes;
  int bot_pe = (mype + npes - 1) % npes;
  constexpr int dim_block_x = 32;
  constexpr int dim_block_y = 32;
  dim3 thread_dim(dim_block_x, dim_block_x);
  dim3 block_dim((N + dim_block_x - 1) / dim_block_x,
                 ((iy_end - iy_start) + dim_block_y - 1) / dim_block_y);
  jacobi_kernel<dim_block_x, dim_block_y><<<block_dim, thread_dim, 0, stream>>>(
      a_new, a, iy_start, iy_end, N, top_pe, bot_pe);
  CUDA_CALL(hipGetLastError());
}

template <int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void jacobi_kernel(real *__restrict__ const a_new, const real *__restrict__ const a,
                              const int iy_start, const int iy_end, const int N, int top_pe, int bot_pe)
{
  int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start;
  int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;

  if (iy < iy_end && ix < (N - 1))
  {
    const real new_val = 0.25 * (a[iy * N + ix + 1] + a[iy * N + ix - 1] +
                                 a[(iy + 1) * N + ix] + a[(iy - 1) * N + ix]);
    a_new[iy * N + ix] = new_val;
    // Halo exchange
    if (iy == iy_start )
    {
      nvshmem_double_p(a_new + (iy * (N-1) + ix), new_val, bot_pe);
    }
    if (iy == iy_end - 1 )
    {
      nvshmem_double_p(a_new  + ix, new_val, top_pe);
    }
  }
}
