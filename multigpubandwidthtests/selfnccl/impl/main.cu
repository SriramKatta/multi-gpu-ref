#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <nccl.h>
#include <math.h>

#define number_of_messages 500
#define number_of_warmups 10

#define CUDA_CHECK(stmt)                                       \
  do                                                           \
  {                                                            \
    hipError_t result = (stmt);                               \
    if (hipSuccess != result)                                 \
    {                                                          \
      fprintf(stderr, "[%s:%d] CUDA failed with %s \n",        \
              __FILE__, __LINE__, hipGetErrorString(result)); \
      exit(-1);                                                \
    }                                                          \
  } while (0)

#define NCCL_CHECK(stmt)                                       \
  do                                                           \
  {                                                            \
    ncclResult_t result = (stmt);                              \
    if (ncclSuccess != result)                                 \
    {                                                          \
      fprintf(stderr, "[%s:%d] NCCL error: %s\n",              \
              __FILE__, __LINE__, ncclGetErrorString(result)); \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  } while (0)

__global__ void simple_shift(float *destination, float *source, size_t N)
{
  const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < N)
  {
    source[idx] += 1.0f;
  }
}

int main(int argc, char *argv[])
{
  MPI_Init(&argc, &argv);
  int rank, nranks;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);

  CUDA_CHECK(hipSetDevice(rank));

  // NCCL Initialization
  ncclComm_t ncclcomm;
  ncclUniqueId id;
  if (rank == 0)
    ncclGetUniqueId(&id);
  MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
  NCCL_CHECK(ncclCommInitRank(&ncclcomm, nranks, id, rank));

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // Benchmark parameters
  size_t base_val = 2;
  size_t max_pow = 32;
  size_t stride_pow = 1;
  int num_median_runs = 5;

  size_t max_size_bytes = (size_t)pow(base_val, max_pow);
  size_t max_num_elements = max_size_bytes / sizeof(float);

  float *host_buff;
  float *device_buff1, *device_buff2;

  CUDA_CHECK(hipHostMalloc(&host_buff, sizeof(float) * max_num_elements));
  CUDA_CHECK(hipMalloc(&device_buff1, sizeof(float) * max_num_elements));
  CUDA_CHECK(hipMalloc(&device_buff2, sizeof(float) * max_num_elements));

  for (size_t i = 0; i < max_num_elements; ++i)
    host_buff[i] = (float)rand() / (float)RAND_MAX;

  CUDA_CHECK(hipMemcpy(device_buff1, host_buff, sizeof(float) * max_num_elements, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(device_buff2, host_buff, sizeof(float) * max_num_elements, hipMemcpyHostToDevice));

  int next = (rank + 1) % nranks;
  int prev = (rank + nranks - 1) % nranks;

  for (size_t pow = 3; pow <= max_pow; pow += stride_pow)
  {
    size_t messagesizebytes = (size_t)powf(base_val, pow);
    size_t num_elements = messagesizebytes / sizeof(float);
    dim3 thpblk(512);
    dim3 numblks((num_elements + thpblk.x - 1) / thpblk.x);

    // if (rank == 0)
    //   printf("\n[%s] Testing message size: %zu^%zu = %zu bytes\n", __TIME__, base_val, pow, messagesizebytes);
    double avgbw = 0.0;
    for (int median_loop = 1; median_loop <= num_median_runs; ++median_loop)
    {
      MPI_Barrier(MPI_COMM_WORLD);

      // 🔥 Warmup
      for (int w = 0; w < number_of_warmups; ++w)
      {
#ifndef NO_COMP
        simple_shift<<<numblks, thpblk, 0, stream>>>(device_buff1, device_buff2, num_elements);
#endif
        NCCL_CHECK(ncclGroupStart());
        NCCL_CHECK(ncclSend(device_buff2, num_elements, ncclFloat, next, ncclcomm, stream));
        NCCL_CHECK(ncclRecv(device_buff1, num_elements, ncclFloat, prev, ncclcomm, stream));
        NCCL_CHECK(ncclGroupEnd());
        std::swap(device_buff1, device_buff2);
      }

      CUDA_CHECK(hipStreamSynchronize(stream));
      MPI_Barrier(MPI_COMM_WORLD);
      double start = MPI_Wtime();

      for (size_t i = 0; i < number_of_messages; i++)
      {
#ifndef NO_COMP
        simple_shift<<<numblks, thpblk, 0, stream>>>(device_buff1, device_buff2, num_elements);
#endif
        NCCL_CHECK(ncclGroupStart());
        NCCL_CHECK(ncclSend(device_buff2, num_elements, ncclFloat, next, ncclcomm, stream));
        NCCL_CHECK(ncclRecv(device_buff1, num_elements, ncclFloat, prev, ncclcomm, stream));
        NCCL_CHECK(ncclGroupEnd());
        std::swap(device_buff1, device_buff2);
      }

      CUDA_CHECK(hipStreamSynchronize(stream));
      double local_dur = MPI_Wtime() - start;
      double max_dur = 0.0;

      MPI_Reduce(&local_dur, &max_dur, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

      if (rank == 0)
      {
        double time_per_msg = (max_dur) / (number_of_messages);
        double bandwidth = (2.0 * messagesizebytes / time_per_msg) / 1e9; // GB/s
        avgbw += bandwidth;
        if (median_loop == num_median_runs)
        {
          avgbw /= static_cast<double>(num_median_runs);
          printf(
              "NP %3d | Message_Size %12zu | avgbw %7.3f GB/s\n",
              nranks,
              messagesizebytes,
              avgbw);
        }
      }
    }
  }

  CUDA_CHECK(hipHostFree(host_buff));
  CUDA_CHECK(hipFree(device_buff1));
  CUDA_CHECK(hipFree(device_buff2));
  CUDA_CHECK(hipStreamDestroy(stream));
  ncclCommDestroy(ncclcomm);
  MPI_Finalize();
  return 0;
}
